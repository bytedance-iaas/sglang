#include "hip/hip_runtime.h"
#include <cudaTypedefs.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

// TESTING
#include "pytorch_extension_utils.h"

#define ENABLE_CUTLASS_MOE_SM90 1 // -> in CMakeLists.txt but doesn't work. TODO: use SGL_KERNEL_ENABLE_SM90A in CMakeLists.txt e.g., make build -DSGL_KERNEL_ENABLE_SM90A. And check SGL_KERNEL_ENABLE_SM90A below.
// #include "cutlass_extensions/common.hpp"
// get_sm_version_num

/* TODO */
// /nvme0n1/jack/vllm-w8a8-cutlass/csrc/quantization/cutlass_w8a8/moe/grouped_mm_c3x.cu
// cutlass_moe_mm_sm90()

/*
HACK
hope this doesn't need common.hpp.....
*/
// #include "cutlass_extensions/common.hpp"

int32_t get_sm_version_num() {
  int32_t major_capability, minor_capability;
  hipDeviceGetAttribute(&major_capability, hipDeviceAttributeComputeCapabilityMajor,
                         0);
  hipDeviceGetAttribute(&minor_capability, hipDeviceAttributeComputeCapabilityMinor,
                         0);
  int32_t version_num = major_capability * 10 + minor_capability;
  return version_num;
}


/*
 HACK
 vllm/csrc/quantization/cutlass_w8a8/moe/moe_data.cu
*/
/*
// #include <cudaTypedefs.h>

// #include <c10/cuda/CUDAGuard.h>
// #include <torch/all.h>

#include <iostream>

constexpr uint64_t THREADS_PER_EXPERT = 512;

__global__ void compute_problem_sizes(const int* __restrict__ topk_ids,
                                      int32_t* problem_sizes1,
                                      int32_t* problem_sizes2,
                                      int32_t* atomic_buffer,
                                      const int topk_length, const int n,
                                      const int k) {
  int expert_id = blockIdx.x;

  int occurrences = 0;
  for (int i = threadIdx.x; i < topk_length; i += THREADS_PER_EXPERT) {
    occurrences += (topk_ids[i] == expert_id);
  }
  atomicAdd(&atomic_buffer[expert_id], occurrences);
  __syncthreads();

  if (threadIdx.x == 0) {
    int final_occurrences = atomic_buffer[expert_id];
    problem_sizes1[expert_id * 3] = final_occurrences;
    problem_sizes1[expert_id * 3 + 1] = 2 * n;
    problem_sizes1[expert_id * 3 + 2] = k;
    problem_sizes2[expert_id * 3] = final_occurrences;
    problem_sizes2[expert_id * 3 + 1] = k;
    problem_sizes2[expert_id * 3 + 2] = n;
  }
}

__global__ void compute_expert_offsets(
    const int32_t* __restrict__ problem_sizes1, int32_t* expert_offsets,
    int32_t* atomic_buffer, const int num_experts) {
  int32_t tot_offset = 0;
  expert_offsets[0] = 0;
  for (int i = 0; i < num_experts; ++i) {
    atomic_buffer[i] = tot_offset;
    tot_offset += problem_sizes1[i * 3];
    expert_offsets[i + 1] = tot_offset;
  }
}

__global__ void compute_arg_sorts(const int* __restrict__ topk_ids,
                                  const int32_t* __restrict__ expert_offsets,
                                  int32_t* input_permutation,
                                  int32_t* output_permutation,
                                  int32_t* atomic_buffer, const int topk_length,
                                  const int topk) {
  int const blk_expert_id = blockIdx.x;
  int const num_experts = gridDim.x;
  int32_t const num_tokens = expert_offsets[num_experts];

  for (int i = threadIdx.x; i < topk_length; i += THREADS_PER_EXPERT) {
    int const expert_id = topk_ids[i];
    if (expert_id == -1 && blockIdx.x == 0) {
      // output_permutation is used to re-order the moe outputs. It is
      // used as c2 = c2[c_map], where c2 is a torch.tensor that is the
      // output of the cutlass kernels and c_map is the output_permutation.
      // c2 is initialized to zeros, therefore by setting the output_permutation
      // to num_tokens, we are guaranteed to fill the moe outputs to zero
      // for "invalid" topk_ids.
      output_permutation[i] = num_tokens;
    } else if (expert_id == blk_expert_id) {
      int start = atomicAdd(&atomic_buffer[expert_id], 1);
      input_permutation[start] = i / topk;
      output_permutation[i] = start;
    }
  }
}


void get_cutlass_moe_mm_data_caller(
  const torch::Tensor& topk_ids, torch::Tensor& expert_offsets,
  torch::Tensor& problem_sizes1, torch::Tensor& problem_sizes2,
  torch::Tensor& input_permutation, torch::Tensor& output_permutation,
  const int64_t num_experts, const int64_t n, const int64_t k) {
auto stream = at::cuda::getCurrentCUDAStream(topk_ids.device().index());
auto options_int32 =
    torch::TensorOptions().dtype(torch::kInt32).device(topk_ids.device());
torch::Tensor atomic_buffer = torch::zeros(num_experts, options_int32);

int num_threads = min(THREADS_PER_EXPERT, topk_ids.numel());
compute_problem_sizes<<<num_experts, num_threads, 0, stream>>>(
    static_cast<const int32_t*>(topk_ids.data_ptr()),
    static_cast<int32_t*>(problem_sizes1.data_ptr()),
    static_cast<int32_t*>(problem_sizes2.data_ptr()),
    static_cast<int32_t*>(atomic_buffer.data_ptr()), topk_ids.numel(), n, k);
compute_expert_offsets<<<1, 1, 0, stream>>>(
    static_cast<const int32_t*>(problem_sizes1.data_ptr()),
    static_cast<int32_t*>(expert_offsets.data_ptr()),
    static_cast<int32_t*>(atomic_buffer.data_ptr()), num_experts);
compute_arg_sorts<<<num_experts, num_threads, 0, stream>>>(
    static_cast<const int32_t*>(topk_ids.data_ptr()),
    static_cast<const int32_t*>(expert_offsets.data_ptr()),
    static_cast<int32_t*>(input_permutation.data_ptr()),
    static_cast<int32_t*>(output_permutation.data_ptr()),
    static_cast<int32_t*>(atomic_buffer.data_ptr()), topk_ids.numel(),
    topk_ids.size(1));
}
*/




/*
void cutlass_scaled_mm_sm75(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            std::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_sm80(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            std::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_sm89(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            std::optional<torch::Tensor> const& bias);
*/

#if defined ENABLE_SCALED_MM_SM90 && ENABLE_SCALED_MM_SM90
/*
void cutlass_scaled_mm_sm90(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            std::optional<torch::Tensor> const& bias);
*/
void cutlass_moe_mm_sm90(
    torch::Tensor& out_tensors, torch::Tensor const& a_tensors,
    torch::Tensor const& b_tensors, torch::Tensor const& a_scales,
    torch::Tensor const& b_scales, torch::Tensor const& expert_offsets,
    torch::Tensor const& problem_sizes, torch::Tensor const& a_strides,
    torch::Tensor const& b_strides, torch::Tensor const& c_strides);

void get_cutlass_moe_mm_data_caller(
    const torch::Tensor& topk_ids, torch::Tensor& expert_offsets,
    torch::Tensor& problem_sizes1, torch::Tensor& problem_sizes2,
    torch::Tensor& input_permutation, torch::Tensor& output_permutation,
    const int64_t num_experts, const int64_t n, const int64_t k);

#endif

/*
#if defined ENABLE_SCALED_MM_SM100 && ENABLE_SCALED_MM_SM100
void cutlass_scaled_mm_sm100(torch::Tensor& c, torch::Tensor const& a,
                             torch::Tensor const& b,
                             torch::Tensor const& a_scales,
                             torch::Tensor const& b_scales,
                             std::optional<torch::Tensor> const& bias);
#endif
*/

/*
void cutlass_scaled_mm_azp_sm75(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                std::optional<torch::Tensor> const& azp,
                                std::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_azp_sm80(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                std::optional<torch::Tensor> const& azp,
                                std::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_azp_sm89(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                std::optional<torch::Tensor> const& azp,
                                std::optional<torch::Tensor> const& bias);
*/

/*
#if defined ENABLE_SCALED_MM_SM90 && ENABLE_SCALED_MM_SM90
void cutlass_scaled_mm_azp_sm90(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                std::optional<torch::Tensor> const& azp,
                                std::optional<torch::Tensor> const& bias);
#endif
*/

/*
bool cutlass_scaled_mm_supports_fp8(int64_t cuda_device_capability) {
  // CUTLASS FP8 kernels need at least
  //   CUDA 12.0 on SM90 systems (Hopper)
  //   CUDA 12.4 on SM89 systems (Lovelace)

#if defined CUDA_VERSION
  if (cuda_device_capability >= 90) {
    return CUDA_VERSION >= 12000;
  } else if (cuda_device_capability >= 89) {
    return CUDA_VERSION >= 12040;
  }
#endif

  return false;
}


bool cutlass_scaled_mm_supports_block_fp8(int64_t cuda_device_capability) {
  // CUTLASS block-quantized FP8 kernels need at least CUDA 12.0
  // and at least SM90 (Hopper)

#if defined CUDA_VERSION
  if (cuda_device_capability >= 90 && cuda_device_capability < 100) {
    return CUDA_VERSION >= 12000;
  }
#endif

  return false;
}
*/

/*
bool cutlass_group_gemm_supported(int64_t cuda_device_capability) {
  // CUTLASS groped FP8 kernels need at least CUDA 12.3
  // and SM90 (Hopper)

#if defined CUDA_VERSION
  if (cuda_device_capability == 90) {
    return CUDA_VERSION >= 12030;
  }
#endif

  return false;
}
*/

/*
void cutlass_scaled_mm(torch::Tensor& c, torch::Tensor const& a,
                       torch::Tensor const& b, torch::Tensor const& a_scales,
                       torch::Tensor const& b_scales,
                       std::optional<torch::Tensor> const& bias) {
  // Checks for conformality
  TORCH_CHECK(a.dim() == 2 && b.dim() == 2 && c.dim() == 2);
  TORCH_CHECK(c.size(0) == a.size(0) && a.size(1) == b.size(0) &&
              b.size(1) == c.size(1));

  // Check for strides and alignment
  TORCH_CHECK(a.stride(1) == 1 && c.stride(1) == 1);  // Row-major
  TORCH_CHECK(b.stride(0) == 1);                      // Column-major
  TORCH_CHECK(c.stride(0) % 16 == 0 &&
              b.stride(1) % 16 == 0);  // 16 Byte Alignment

  if (bias) {
    TORCH_CHECK(bias->numel() == b.size(1) && bias->is_contiguous() &&
                bias->dim() == 1);
  }

  at::cuda::OptionalCUDAGuard const device_guard(device_of(a));
  int32_t version_num = get_sm_version_num();

#if defined ENABLE_SCALED_MM_SM100 && ENABLE_SCALED_MM_SM100
  if (version_num >= 100) {
    cutlass_scaled_mm_sm100(c, a, b, a_scales, b_scales, bias);
    return;
  }
#endif

  // Guard against compilation issues for sm90 kernels
#if defined ENABLE_SCALED_MM_SM90 && ENABLE_SCALED_MM_SM90
  if (version_num >= 90 && version_num < 100) {
    // Hopper
    cutlass_scaled_mm_sm90(c, a, b, a_scales, b_scales, bias);
    return;
  }
#endif

#if defined ENABLE_SCALED_MM_C2X && ENABLE_SCALED_MM_C2X
  if (version_num == 89) {
    // Ada Lovelace
    cutlass_scaled_mm_sm89(c, a, b, a_scales, b_scales, bias);
    return;
  }

  if (version_num >= 80) {
    // Ampere
    cutlass_scaled_mm_sm80(c, a, b, a_scales, b_scales, bias);
    return;
  }

  if (version_num >= 75) {
    // Turing
    cutlass_scaled_mm_sm75(c, a, b, a_scales, b_scales, bias);
    return;
  }
#endif

  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled cutlass_scaled_mm for a compute capability less than "
      "CUDA device capability: ",
      version_num);
}
*/

void cutlass_moe_mm(
    torch::Tensor& out_tensors, torch::Tensor const& a_tensors,
    torch::Tensor const& b_tensors, torch::Tensor const& a_scales,
    torch::Tensor const& b_scales, torch::Tensor const& expert_offsets,
    torch::Tensor const& problem_sizes, torch::Tensor const& a_strides,
    torch::Tensor const& b_strides, torch::Tensor const& c_strides) {
  int32_t version_num = get_sm_version_num();
#if defined ENABLE_CUTLASS_MOE_SM90 && ENABLE_CUTLASS_MOE_SM90
  cutlass_moe_mm_sm90(out_tensors, a_tensors, b_tensors, a_scales, b_scales,
                      expert_offsets, problem_sizes, a_strides, b_strides,
                      c_strides);
  return;
#endif
  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled cutlass_scaled_mm for CUDA device capability: ", version_num,
      ". Required capability: 90");
}

void get_cutlass_moe_mm_data(
    const torch::Tensor& topk_ids, torch::Tensor& expert_offsets,
    torch::Tensor& problem_sizes1, torch::Tensor& problem_sizes2,
    torch::Tensor& input_permutation, torch::Tensor& output_permutation,
    const int64_t num_experts, const int64_t n, const int64_t k) {
  // This function currently gets compiled only if we have a valid cutlass moe
  // mm to run it for.
#if defined ENABLE_CUTLASS_MOE_SM90
  #if 0
    printf("Jack get_cutlass_moe_mm_data(): torch_binding working defined ENABLE_CUTLASS_MOE_SM90 %d\n", ENABLE_CUTLASS_MOE_SM90);
  #endif
#else
  #if 0:
    printf("Jack get_cutlass_moe_mm_data(): torch_binding working not defined ENABLE_CUTLASS_MOE_SM90\n");
  #endif
#endif
  int32_t version_num = get_sm_version_num();
#if defined ENABLE_CUTLASS_MOE_SM90 && ENABLE_CUTLASS_MOE_SM90
  get_cutlass_moe_mm_data_caller(topk_ids, expert_offsets, problem_sizes1,
                                 problem_sizes2, input_permutation,
                                 output_permutation, num_experts, n, k);
  return;
#endif
  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled get_cutlass_moe_mm_data: no cutlass_scaled_mm kernel for "
      "CUDA device capability: ",
      version_num, ". Required capability: 90");
}

/*
void cutlass_scaled_mm_azp(torch::Tensor& c, torch::Tensor const& a,
                           torch::Tensor const& b,
                           torch::Tensor const& a_scales,
                           torch::Tensor const& b_scales,
                           torch::Tensor const& azp_adj,
                           std::optional<torch::Tensor> const& azp,
                           std::optional<torch::Tensor> const& bias) {
  // Checks for conformality
  TORCH_CHECK(a.dim() == 2 && b.dim() == 2 && c.dim() == 2);
  TORCH_CHECK(c.size(0) == a.size(0) && a.size(1) == b.size(0) &&
              b.size(1) == c.size(1));
  TORCH_CHECK(a_scales.numel() == 1 || a_scales.numel() == a.size(0));
  TORCH_CHECK(b_scales.numel() == 1 || b_scales.numel() == b.size(1));

  // Check for strides and alignment
  TORCH_CHECK(a.stride(1) == 1 && c.stride(1) == 1);  // Row-major
  TORCH_CHECK(b.stride(0) == 1);                      // Column-major
  TORCH_CHECK(c.stride(0) % 16 == 0 &&
              b.stride(1) % 16 == 0);  // 16 Byte Alignment
  TORCH_CHECK(a_scales.is_contiguous() && b_scales.is_contiguous());

  // bias, azp, azp_adj are all 1d
  // bias and azp_adj have n elements, azp has m elements
  if (bias) {
    TORCH_CHECK(bias->numel() == b.size(1) && bias->is_contiguous());
  }
  if (azp) {
    TORCH_CHECK(azp->numel() == a.size(0) && azp->is_contiguous());
  }
  TORCH_CHECK(azp_adj.numel() == b.size(1) && azp_adj.is_contiguous());

  // azp & bias types
  TORCH_CHECK(azp_adj.dtype() == torch::kInt32);
  TORCH_CHECK(!azp || azp->dtype() == torch::kInt32);
  TORCH_CHECK(!bias || bias->dtype() == c.dtype(),
              "currently bias dtype must match output dtype ", c.dtype());

  at::cuda::OptionalCUDAGuard const device_guard(device_of(a));

  int32_t version_num = get_sm_version_num();

#if defined ENABLE_SCALED_MM_SM90 && ENABLE_SCALED_MM_SM90
  if (version_num >= 90) {
    cutlass_scaled_mm_azp_sm90(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
    return;
  }
#endif

#if defined ENABLE_SCALED_MM_C2X && ENABLE_SCALED_MM_C2X
  if (version_num == 89) {
    // Ada Lovelace
    cutlass_scaled_mm_azp_sm89(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
    return;
  }

  if (version_num >= 80) {
    // Ampere
    cutlass_scaled_mm_azp_sm80(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
    return;
  }

  // Turing
  TORCH_CHECK(version_num >= 75);
  cutlass_scaled_mm_azp_sm75(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
  return;
#endif

  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled cutlass_scaled_mm_azp for a compute capability less than "
      "CUDA device capability: ",
      version_num);
}

*/