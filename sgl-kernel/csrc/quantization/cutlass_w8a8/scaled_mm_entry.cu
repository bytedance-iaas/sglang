#include <cudaTypedefs.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

int32_t get_sm_version_num() {
  int32_t major_capability, minor_capability;
  hipDeviceGetAttribute(&major_capability, hipDeviceAttributeComputeCapabilityMajor,
                         0);
  hipDeviceGetAttribute(&minor_capability, hipDeviceAttributeComputeCapabilityMinor,
                         0);
  int32_t version_num = major_capability * 10 + minor_capability;
  return version_num;
}


#if defined ENABLE_SCALED_MM_SM90 && ENABLE_SCALED_MM_SM90
/*
void cutlass_scaled_mm_sm90(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            std::optional<torch::Tensor> const& bias);
*/
void cutlass_moe_mm_sm90(
    torch::Tensor& out_tensors, torch::Tensor const& a_tensors,
    torch::Tensor const& b_tensors, torch::Tensor const& a_scales,
    torch::Tensor const& b_scales, torch::Tensor const& expert_offsets,
    torch::Tensor const& problem_sizes, torch::Tensor const& a_strides,
    torch::Tensor const& b_strides, torch::Tensor const& c_strides);

void get_cutlass_moe_mm_data_caller(
    const torch::Tensor& topk_ids, torch::Tensor& expert_offsets,
    torch::Tensor& problem_sizes1, torch::Tensor& problem_sizes2,
    torch::Tensor& input_permutation, torch::Tensor& output_permutation,
    const int64_t num_experts, const int64_t n, const int64_t k);

#endif


void cutlass_moe_mm(
    torch::Tensor& out_tensors, torch::Tensor const& a_tensors,
    torch::Tensor const& b_tensors, torch::Tensor const& a_scales,
    torch::Tensor const& b_scales, torch::Tensor const& expert_offsets,
    torch::Tensor const& problem_sizes, torch::Tensor const& a_strides,
    torch::Tensor const& b_strides, torch::Tensor const& c_strides) {
  int32_t version_num = get_sm_version_num();
#if defined ENABLE_CUTLASS_MOE_SM90 && ENABLE_CUTLASS_MOE_SM90
// #if defined SGL_KERNEL_ENABLE_SM90A && SGL_KERNEL_ENABLE_SM90A
  cutlass_moe_mm_sm90(out_tensors, a_tensors, b_tensors, a_scales, b_scales,
                      expert_offsets, problem_sizes, a_strides, b_strides,
                      c_strides);
  return;
#endif
  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled cutlass_scaled_mm for CUDA device capability: ", version_num,
      ". Required capability: 90");
}

void get_cutlass_moe_mm_data(
    const torch::Tensor& topk_ids, torch::Tensor& expert_offsets,
    torch::Tensor& problem_sizes1, torch::Tensor& problem_sizes2,
    torch::Tensor& input_permutation, torch::Tensor& output_permutation,
    const int64_t num_experts, const int64_t n, const int64_t k) {
  // This function currently gets compiled only if we have a valid cutlass moe
  // mm to run it for.
  int32_t version_num = get_sm_version_num();
#if defined ENABLE_CUTLASS_MOE_SM90 && ENABLE_CUTLASS_MOE_SM90
  get_cutlass_moe_mm_data_caller(topk_ids, expert_offsets, problem_sizes1,
                                 problem_sizes2, input_permutation,
                                 output_permutation, num_experts, n, k);
  return;
#endif
  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled get_cutlass_moe_mm_data: no cutlass_scaled_mm kernel for "
      "CUDA device capability: ",
      version_num, ". Required capability: 90");
}
