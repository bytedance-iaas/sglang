#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <THC/THCAtomics.cuh>

#include "utils.h"

__global__ void ep_pre_reorder_cuda_kernel(
    const float* __restrict__ input_ptr,
    float* __restrict__ gateup_input_ptr,
    const int* __restrict__ src2dst_ptr,
    const int* __restrict__ topk_ids_ptr,
    const float* __restrict__ a1_scales_ptr,
    int start_expert_id,
    int end_expert_id,
    int topk,
    int hidden_size) {
  int token_idx = blockIdx.x;
  int tid = threadIdx.x;

  const float* src_ptr = input_ptr + token_idx * hidden_size;
  const int* token_src2dst = src2dst_ptr + token_idx * topk;
  const int* token_topk_ids = topk_ids_ptr + token_idx * topk;

  for (int k = 0; k < topk; ++k) {
    int expert_id = token_topk_ids[k];
    if (expert_id < start_expert_id || expert_id > end_expert_id) continue;

    float scale = 1.0f;
    if (a1_scales_ptr != nullptr) {
      scale = 1.0f / a1_scales_ptr[expert_id - start_expert_id];
    }

    int dst_idx = token_src2dst[k];
    float* dst_ptr = gateup_input_ptr + dst_idx * hidden_size;

    for (int i = tid; i < hidden_size; i += blockDim.x) {
      dst_ptr[i] = src_ptr[i] * scale;
    }
  }
}

void ep_moe_pre_reorder(
    torch::Tensor input,
    torch::Tensor gateup_input,
    torch::Tensor src2dst,
    torch::Tensor topk_ids,
    torch::Tensor a1_scales,
    int64_t start_expert_id,
    int64_t end_expert_id,
    int64_t topk) {
  int total_blocks = input.size(0);
  int block_size = 512;
  dim3 grid(total_blocks);
  dim3 block(block_size);
  int hidden_size = input.size(1);
  ep_pre_reorder_cuda_kernel<<<grid, block>>>(
      input.data_ptr<float>(),
      gateup_input.data_ptr<float>(),
      src2dst.data_ptr<int>(),
      topk_ids.data_ptr<int>(),
      a1_scales.defined() ? a1_scales.data_ptr<float>() : nullptr,
      start_expert_id,
      end_expert_id,
      topk,
      hidden_size);
}
